#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdarg.h>
#include <errno.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <time.h>
#include <unistd.h>

#define minBYpH 16
#define BYpH 64L
#define HpT 50000L
#define TpB 128L
#define BLOCKS 15L

#define NO_MAIN
#define GPU
#include "threefish.c"

__global__
void search(uint8_t* rands, int* ans)
{
	const uint64_t target[16] = {0x8082a05f5fa94d5b,0xc818f444df7998fc,0x7d75b724a42bf1f9,0x4f4c0daefbbd2be0,0x04fec50cc81793df,0x97f26c46739042c6,0xf6d2dd9959c2b806,0x877b97cc75440d54,0x8f9bf123e07b75f4,0x88b7862872d73540,0xf99ca716e96d8269,0x247d34d49cc74cc9,0x73a590233eaa67b5,0x4066675e8aa473a3,0xe7c5e19701c79cc7,0xb65818ca53fb02f9};
	uint64_t hash[16];

	int min = 1024;

	for(int i = 0; i < HpT; i++)
	{
		skeinhash1024x1024(rands+blockIdx.x*TpB*HpT*BYpH+threadIdx.x*HpT*BYpH+i*BYpH, BYpH, hash);
		int c = 0;
		for (int j = 0; j < Nw; j++)
		{
			uint64_t tmp = target[j] ^ hash[j];
			for (int k = 0; k < 64; k++)
			{
				if(tmp & 1)
					c++;
				tmp >>= 1;
			}
		}
		if(c < min)
		{
			min = c;
		}
	}

	ans[blockIdx.x*TpB+threadIdx.x] = min;
}

void printLog(char* msg);
void printfLog(char* fmt, ...);

int main()
{
	// uint8_t* rands;
	int* answers;
	uint8_t* cuRands;
	int* cuAns;
	hiprandStatus_t s;

	// Asserts on constants
	assert(BYpH%sizeof(int) == 0); // cuRand generator makes ints
	assert((BLOCKS*TpB)%2 == 0); //

	// rands = (uint8_t*) malloc(TpB*HpT*BYpH); assert(rands != NULL);
	answers = (int*) malloc(BLOCKS*TpB*sizeof(int)); assert(answers != NULL);

	hipSetDeviceFlags(hipDeviceScheduleBlockingSync); assert(hipGetLastError() == hipSuccess);

	hipMalloc((void**)&cuRands, BLOCKS*TpB*HpT*BYpH); assert(hipGetLastError() == hipSuccess);
	hipMalloc((void**)&cuAns, BLOCKS*TpB*sizeof(int)); assert(hipGetLastError() == hipSuccess);
	hipMemset(cuAns, 0, BLOCKS*TpB*sizeof(int)); assert(hipGetLastError() == hipSuccess);

	// printfLog("cuRands size: %d", BLOCKS*TpB*HpT*BYpH);
	// printfLog("rands size: %d", TpB*HpT*BYpH);

	FILE* fp = fopen("/dev/urandom", "r");
	uint64_t seed;
	int ret = fread(&seed, sizeof(uint64_t), 1, fp);

	printLog((char*)"generating/transferring random data for computation");

	hiprandGenerator_t gen;
	s = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_XORWOW); assert(s == HIPRAND_STATUS_SUCCESS);
	s = hiprandSetPseudoRandomGeneratorSeed(gen,seed); assert(s == HIPRAND_STATUS_SUCCESS);
	s = hiprandGenerate(gen, (unsigned int*) cuRands, BYpH/4*HpT*TpB*BLOCKS); assert(s == HIPRAND_STATUS_SUCCESS);
	hipDeviceSynchronize(); assert(hipGetLastError() == hipSuccess);
	s = hiprandDestroyGenerator(gen); assert(s == HIPRAND_STATUS_SUCCESS);

	// for(int i = 0; i < BLOCKS; i++)
	// {
	// 	int ret = fread(rands, BYpH, TpB*HpT, fp);
	// 	hipMemcpy(cuRands+i*TpB*HpT*BYpH, rands, TpB*HpT*BYpH, hipMemcpyHostToDevice); assert(hipGetLastError() == hipSuccess);
	// 	printfLog("% 6.2f%% complete", (double)i/(double)BLOCKS*100.0);
	// }
	fclose(fp);
	
	printLog((char*)"starting search");
	search<<<BLOCKS, TpB>>>(cuRands, cuAns); assert(hipGetLastError() == hipSuccess);
	hipDeviceSynchronize(); assert(hipGetLastError() == hipSuccess);
	printLog((char*)"search finished");

	hipMemcpy(answers, cuAns, BLOCKS*TpB*sizeof(int), hipMemcpyDeviceToHost); assert(hipGetLastError() == hipSuccess);
	hipFree(cuAns); assert(hipGetLastError() == hipSuccess);

	printLog((char*)"finding best match");
	int min = 1024;
	int lowestI = -1;
	for(int i = 0; i < BLOCKS*TpB; i++)
	{
		if(answers[i] < min)
		{
			min = answers[i];
			lowestI = i;
		}
	}

	uint8_t bestMatch[BYpH];
	hipMemcpy(&bestMatch, cuRands+lowestI*BYpH, BYpH, hipMemcpyDeviceToHost); assert(hipGetLastError() == hipSuccess);
	printfLog((char*)"best match(%d incorrect bits):", min);
	for(int i = 0; i < BYpH; i++)
	{
		printf("%02x ", bestMatch[i]);
	}
	printf("\n");
	hipFree(cuRands); assert(hipGetLastError() == hipSuccess);

	return EXIT_SUCCESS;
}

void printLog(char* msg)
{
	time_t now;
	struct tm* 	lcltime;

	now = time(NULL);
	lcltime = localtime(&now);
	FILE* fp = stdout;
	fprintf(fp, "%d-%02d-%02d %02d:%02d:%02d ~ %s\n", lcltime->tm_year + 1900, lcltime->tm_mon + 1, lcltime->tm_mday, lcltime->tm_hour, lcltime->tm_min, lcltime->tm_sec, msg);
	fflush(stdout);
}

void printfLog(char* fmt, ...)
{
	char* msg;

	va_list ap;
	va_start(ap, fmt);

	msg = (char*) malloc(vsnprintf(NULL, 0, fmt, ap) + 1);
	va_end(ap);
	if(!msg)
	{
		printLog(strerror(errno));
		return;
	}

	va_start(ap, fmt); // The vsnprintf call clobbered my va_list.  So starting it again.

	vsprintf(msg, fmt, ap);
	printLog(msg);

	free(msg);
	va_end(ap);
}
